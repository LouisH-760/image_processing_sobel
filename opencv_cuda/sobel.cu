#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>

#define WINDOWFLAGS WINDOW_NORMAL|WINDOW_KEEPRATIO|WINDOW_GUI_EXPANDED
#define WNAME "Sobel"
#define OUTNAME "result.png"
#define DISPLAY_SCALE 0.2

#define BLOCK_SIZE 32


using namespace cv;

int showAndSave(Mat sobel)
{
    Mat resized;
    resize(sobel, resized, Size(), DISPLAY_SCALE, DISPLAY_SCALE, INTER_AREA);
    namedWindow(WNAME, WINDOWFLAGS);
    imshow(WNAME, resized);
    imwrite(OUTNAME, sobel);
    return waitKey(0);
}

Mat loadImage(int argc, char** argv) {
    Mat image;
    if (argc != 2) {
        printf("usage: DisplayImage.out <Image_Path>\n");
        return image;
    }
    image = imread( argv[1], IMREAD_GRAYSCALE);
    if (image.empty()) {
        printf("No image data \n");
        return image;
    }
    return image;
}

void matrixToArray(Mat matrix, int* arr, int size) {
    for(int i = 0; i < size; i++) {
        *(arr + i) = (int) matrix.at<uchar>(i%matrix.cols, (int) round(i/matrix.cols));
    }
}

Mat arrayToMatrix(int* arr, int size, int w, int h) {
    Mat out(w, h, CV_8UC1);
    for(int i = 0; i < size; i++) {
        out.at<uchar>(i%w, (int) round(i/w)) = (unsigned char) *(arr + i);
    }
    return out;
}

__global__ void sobelNaive(int *img, int *output, int size) {
    int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if(index < size) {
        int val = *(img + index);
        *(output + index) = val;
    }
}

int main(int argc, char** argv ) {
    printf("begin");
    Mat orig = loadImage(argc, argv);
    printf("loadimg");
    const int dims = orig.cols * orig.rows;
    const int blocks = (int) ceil(dims / BLOCK_SIZE);
    // with a big image, dims > size_t. Can't use arrays :(
    int *img = (int *) malloc(dims * sizeof(int));
    int *remoteImg, *remoteOutput, *output;
    printf("vars");
    hipMalloc(&remoteImg, sizeof(int) * orig.total());
    hipMalloc(&remoteOutput, sizeof(int) *  orig.total());
    printf("allocs");
    // since we're on GPU, we don't want the Mat type but an int array, if possible
    matrixToArray(orig, img,  orig.total());
    printf("init");

    hipMemcpy(remoteImg, img,  orig.total(), hipMemcpyHostToDevice);
    free(img);

    sobelNaive<<<blocks, BLOCK_SIZE>>>(remoteImg, remoteOutput,  orig.total());

    hipDeviceSynchronize();
    hipFree(remoteImg);
    output = (int *) malloc( dims * (sizeof(int)));
    hipMemcpy(output, remoteOutput, sizeof(int) *  orig.total(), hipMemcpyDeviceToHost);

    showAndSave(arrayToMatrix(output, orig.total(), orig.rows, orig.cols));
    
    free(output);
    hipFree(remoteOutput);
    return 0;
}